#include "hip/hip_runtime.h"
#include "iostream"
#include "long_march.h"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"

using namespace long_march;

__global__ void kernel(geometry::Triangle3d *triangles) {
  geometry::Triangle<double, 3> triangle;
  triangle[0] << 1, 0, 0;
  triangle[1] << 0, 1, 0;
  triangle[2] << 0, 0, 1;
  triangles[0] = triangle;
  auto v = triangle.normal();
  printf("%f %f %f\n", v[0], v[1], v[2]);
}

int main() {
  thrust::device_vector<geometry::Triangle3d> triangles(1);
  kernel<<<1, 1>>>(thrust::raw_pointer_cast(triangles.data()));
  hipDeviceSynchronize();
  thrust::host_vector<geometry::Triangle3d> h_triangles = triangles;
  std::cout << h_triangles[0].m << std::endl;
}
