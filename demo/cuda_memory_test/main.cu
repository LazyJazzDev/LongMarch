#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "iostream"
#include "thrust/device_vector.h"

__global__ void MemoryTestKernel() {
  printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

template <int block_size>
struct MemoryBlock {
  uint32_t data[block_size >> 2]{};
  __host__ __device__ MemoryBlock() = default;
  __host__ __device__ MemoryBlock<block_size> operator+(
      const MemoryBlock<block_size> &other) const {
    MemoryBlock<block_size> result;
#pragma unroll
    for (int i = 0; i < block_size >> 2; i++) {
      result.data[i] = data[i] + other.data[i];
    }
    return result;
  }

  __host__ __device__ MemoryBlock<block_size> operator-(
      const MemoryBlock<block_size> &other) const {
    MemoryBlock<block_size> result;
#pragma unroll
    for (int i = 0; i < block_size >> 2; i++) {
      result.data[i] = data[i] - other.data[i];
    }
    return result;
  }

  __host__ __device__ MemoryBlock<block_size> operator*(
      const MemoryBlock<block_size> &other) const {
    MemoryBlock<block_size> result;
#pragma unroll
    for (int i = 0; i < block_size >> 2; i++) {
      result.data[i] = data[i] * other.data[i];
    }
    return result;
  }

  __host__ __device__ MemoryBlock<block_size> operator^(
      const MemoryBlock<block_size> &other) const {
    MemoryBlock<block_size> result;
#pragma unroll
    for (int i = 0; i < block_size >> 2; i++) {
      result.data[i] = data[i] ^ other.data[i];
    }
    return result;
  }

  __host__ __device__ MemoryBlock<block_size> operator+=(
      const MemoryBlock<block_size> &other) {
#pragma unroll
    for (int i = 0; i < block_size >> 2; i++) {
      data[i] += other.data[i];
    }
    return *this;
  }

  __host__ __device__ MemoryBlock<block_size> operator-=(
      const MemoryBlock<block_size> &other) {
#pragma unroll
    for (int i = 0; i < block_size >> 2; i++) {
      data[i] -= other.data[i];
    }
    return *this;
  }

  __host__ __device__ MemoryBlock<block_size> operator*=(
      const MemoryBlock<block_size> &other) {
#pragma unroll
    for (int i = 0; i < block_size >> 2; i++) {
      data[i] *= other.data[i];
    }
    return *this;
  }

  __host__ __device__ MemoryBlock<block_size> operator^=(
      const MemoryBlock<block_size> &other) {
#pragma unroll
    for (int i = 0; i < block_size >> 2; i++) {
      data[i] ^= other.data[i];
    }
    return *this;
  }

  __host__ __device__ bool operator==(
      const MemoryBlock<block_size> &other) const {
#pragma unroll
    for (int i = 0; i < block_size >> 2; i++) {
      if (data[i] != other.data[i]) {
        return false;
      }
    }
    return true;
  }
};

template <int block_size>
__global__ void GenerateRandomBlocksKernel(MemoryBlock<block_size> *data,
                                           int num_elements) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_elements) {
    return;
  }
  hiprandState state{};
  hiprand_init(0, idx, 0, &state);
  for (int i = 0; i < block_size >> 2; i++) {
    data[idx].data[i] = hiprand(&state);
  }
}

template <int block_size>
__global__ void TestRandomAccessThroughputKernel(
    const MemoryBlock<block_size> *data,
    MemoryBlock<block_size> *result,
    int num_elements,
    int num_access) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_elements) {
    return;
  }
  MemoryBlock<block_size> sum{};
  hiprandState state{};
  hiprand_init(0, idx, 0, &state);

#pragma unroll
  for (int i = 0; i < num_access; i++) {
    // rand a int in [0, num_elements - 1]
    uint32_t idy = hiprand(&state) % num_elements;
    MemoryBlock<block_size> other = data[(idx + idy) % num_elements];
    sum ^= other;
  }
  result[idx] = sum;
}

template <int block_size>
void TestRandomAccessThroughput(int num_elements, int num_access) {
  uint64_t total_memory_load = static_cast<uint64_t>(num_elements) *
                               sizeof(MemoryBlock<block_size>) * num_access;
  std::cout << "========================================================\n";
  std::cout << "Testing Random Access - Block Size "
            << sizeof(MemoryBlock<block_size>) << " Num Elements "
            << num_elements << " Num Access " << num_access << std::endl;
  thrust::device_vector<MemoryBlock<block_size>> data(num_elements);
  thrust::device_vector<MemoryBlock<block_size>> result(num_elements);
  float max_throughput = 0;
  float min_throughput = 0;
  float sum_throughput = 0;
  float elapsed_time = 0;
  for (int i = 0; i < 5; i++) {
    GenerateRandomBlocksKernel<block_size><<<num_elements / 256 + 1, 256>>>(
        thrust::raw_pointer_cast(data.data()), num_elements);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    hipEventRecord(start);
    TestRandomAccessThroughputKernel<block_size>
        <<<num_elements / 256 + 1, 256>>>(
            thrust::raw_pointer_cast(data.data()),
            thrust::raw_pointer_cast(result.data()), num_elements, num_access);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    elapsed_time /= 1000;
    float throughput = static_cast<float>(total_memory_load) / elapsed_time /
                       1024 / 1024 / 1024;
    if (i == 0) {
      max_throughput = throughput;
      min_throughput = throughput;
    } else {
      max_throughput = std::max(max_throughput, throughput);
      min_throughput = std::min(min_throughput, throughput);
    }
    sum_throughput += throughput;
  }
  std::cout << "Throughput: max - " << max_throughput << "GB/s min - "
            << min_throughput << "GB/s mean - " << sum_throughput / 5 << "GB/s"
            << std::endl;
}

template <int block_size>
__global__ void TestSequentialAccessThroughputKernel(
    const MemoryBlock<block_size> *data,
    MemoryBlock<block_size> *result,
    int num_elements,
    int num_access) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_elements) {
    return;
  }
  MemoryBlock<block_size> sum{};
  hiprandState state{};
  hiprand_init(0, idx, 0, &state);
  uint32_t idy = hiprand(&state) % num_elements;

#pragma unroll
  for (int i = 0; i < num_access; i++) {
    // rand a int in [0, num_elements - 1]
    MemoryBlock<block_size> other = data[(idx ^ i) % num_elements];
    sum ^= other;
  }
  result[idx] = sum;
}

template <int block_size>
void TestSequentialAccessThroughput(int num_elements, int num_access) {
  uint64_t total_memory_load = static_cast<uint64_t>(num_elements) *
                               sizeof(MemoryBlock<block_size>) * num_access;
  std::cout << "========================================================\n";
  std::cout << "Testing Sequential Access - Block Size "
            << sizeof(MemoryBlock<block_size>) << " Num Elements "
            << num_elements << " Num Access " << num_access << std::endl;
  thrust::device_vector<MemoryBlock<block_size>> data(num_elements);
  thrust::device_vector<MemoryBlock<block_size>> result(num_elements);
  float max_throughput = 0;
  float min_throughput = 0;
  float sum_throughput = 0;
  float elapsed_time = 0;
  for (int i = 0; i < 5; i++) {
    GenerateRandomBlocksKernel<block_size><<<num_elements / 256 + 1, 256>>>(
        thrust::raw_pointer_cast(data.data()), num_elements);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    hipEventRecord(start);
    TestSequentialAccessThroughputKernel<block_size>
        <<<num_elements / 256 + 1, 256>>>(
            thrust::raw_pointer_cast(data.data()),
            thrust::raw_pointer_cast(result.data()), num_elements, num_access);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    elapsed_time /= 1000;
    float throughput = static_cast<float>(total_memory_load) / elapsed_time /
                       1024 / 1024 / 1024;
    if (i == 0) {
      max_throughput = throughput;
      min_throughput = throughput;
    } else {
      max_throughput = std::max(max_throughput, throughput);
      min_throughput = std::min(min_throughput, throughput);
    }
    sum_throughput += throughput;
  }
  std::cout << "Throughput: max - " << max_throughput << "GB/s min - "
            << min_throughput << "GB/s mean - " << sum_throughput / 5 << "GB/s"
            << std::endl;
}

int main() {
  TestRandomAccessThroughput<8>(1024 * 1024, 8192);
  TestRandomAccessThroughput<16>(1024 * 1024, 4096);
  TestRandomAccessThroughput<32>(1024 * 1024, 2048);
  TestRandomAccessThroughput<64>(1024 * 1024, 1024);
  TestRandomAccessThroughput<128>(1024 * 1024, 512);
  TestRandomAccessThroughput<256>(1024 * 1024, 256);
  TestRandomAccessThroughput<512>(1024 * 1024, 128);
  TestRandomAccessThroughput<1024>(1024 * 1024, 64);
  TestRandomAccessThroughput<8>(1024 * 1024 * 128, 64);
  TestRandomAccessThroughput<16>(1024 * 1024 * 64, 64);
  TestRandomAccessThroughput<32>(1024 * 1024 * 32, 64);
  TestRandomAccessThroughput<64>(1024 * 1024 * 16, 64);
  TestRandomAccessThroughput<128>(1024 * 1024 * 8, 64);
  TestRandomAccessThroughput<256>(1024 * 1024 * 4, 64);
  TestRandomAccessThroughput<512>(1024 * 1024 * 2, 64);
  TestRandomAccessThroughput<1024>(1024 * 1024, 64);

  TestSequentialAccessThroughput<8>(1024 * 1024, 8192);
  TestSequentialAccessThroughput<16>(1024 * 1024, 4096);
  TestSequentialAccessThroughput<32>(1024 * 1024, 2048);
  TestSequentialAccessThroughput<64>(1024 * 1024, 1024);
  TestSequentialAccessThroughput<128>(1024 * 1024, 512);
  TestSequentialAccessThroughput<256>(1024 * 1024, 256);
  TestSequentialAccessThroughput<512>(1024 * 1024, 128);
  TestSequentialAccessThroughput<1024>(1024 * 1024, 64);
  TestSequentialAccessThroughput<8>(1024 * 1024 * 128, 64);
  TestSequentialAccessThroughput<16>(1024 * 1024 * 64, 64);
  TestSequentialAccessThroughput<32>(1024 * 1024 * 32, 64);
  TestSequentialAccessThroughput<64>(1024 * 1024 * 16, 64);
  TestSequentialAccessThroughput<128>(1024 * 1024 * 8, 64);
  TestSequentialAccessThroughput<256>(1024 * 1024 * 4, 64);
  TestSequentialAccessThroughput<512>(1024 * 1024 * 2, 64);
  TestSequentialAccessThroughput<1024>(1024 * 1024, 64);
  return 0;
}
