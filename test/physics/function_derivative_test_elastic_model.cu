﻿#include "function_derivative_test.h"

TEST(Physics, FunctionDerivativeElasticNeoHookean) {
  TestFunctionSet<ElasticNeoHookean<double>>();
}

TEST(Physics, FunctionDerivativeElasticNeoHookeanF3x2) {
  TestFunctionSet<ElasticNeoHookeanF3x2<double>>();
}

TEST(Physics, FunctionDerivativeElasticNeoHookeanTriangle) {
  Eigen::Matrix2<double> Dm;
  do {
    Dm = Eigen::Matrix2<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<ElasticNeoHookeanTriangle<double>>({1.0, 1.0, Dm});
}
