#include "hip/hip_runtime.h"
﻿#include <glm/ext/scalar_constants.hpp>
#include <glm/glm.hpp>

#include "cmath"
#include "grassland/physics/physics.h"
#include "gtest/gtest.h"
#include "iostream"
#include "random"

using namespace grassland;

template <typename FunctionSet>
__global__ void DeviceValueKernel(FunctionSet f,
                                  const typename FunctionSet::InputType v,
                                  typename FunctionSet::OutputType *out) {
  out[0] = f(v);
}

template <typename FunctionSet>
__global__ void DeviceJacobianKernel(FunctionSet f,
                                     const typename FunctionSet::InputType v,
                                     JacobianType<FunctionSet> *out) {
  out[0] = f.Jacobian(v);
}

template <typename FunctionSet>
__global__ void DeviceHessianKernel(FunctionSet f,
                                    const typename FunctionSet::InputType v,
                                    HessianType<FunctionSet> *out) {
  out[0] = f.Hessian(v);
}

template <typename FunctionSet>
typename FunctionSet::OutputType DeviceValue(
    FunctionSet f,
    const typename FunctionSet::InputType &x) {
  typename FunctionSet::OutputType *out;
  hipMallocManaged(&out, sizeof(FunctionSet::OutputType));
  DeviceValueKernel<<<1, 1>>>(f, x, out);
  hipDeviceSynchronize();
  typename FunctionSet::OutputType result = *out;
  hipFree(out);
  return result;
}

template <typename FunctionSet>
JacobianType<FunctionSet> DeviceJacobian(
    FunctionSet f,
    const typename FunctionSet::InputType &x) {
  JacobianType<FunctionSet> *out;
  hipMallocManaged(&out, sizeof(JacobianType<FunctionSet>));
  DeviceJacobianKernel<<<1, 1>>>(f, x, out);
  hipDeviceSynchronize();
  JacobianType<FunctionSet> result = *out;
  hipFree(out);
  return result;
}

template <typename FunctionSet>
HessianType<FunctionSet> DeviceHessian(
    FunctionSet f,
    const typename FunctionSet::InputType &x) {
  HessianType<FunctionSet> *out;
  hipMallocManaged(&out, sizeof(HessianType<FunctionSet>));
  DeviceHessianKernel<<<1, 1>>>(f, x, out);
  hipDeviceSynchronize();
  HessianType<FunctionSet> result = *out;
  hipFree(out);
  return result;
}

template <typename FunctionSet = Determinant3<double>>
void TestFunctionSet(FunctionSet f = FunctionSet{}, const int test_cnt = 100) {
  using Real = typename FunctionSet::Scalar;
  for (int i = 0; i < test_cnt; i++) {
    using InputVec =
        Eigen::Vector<Real, FunctionSet::InputType::SizeAtCompileTime>;
    using OutputVec =
        Eigen::Vector<Real, FunctionSet::OutputType::SizeAtCompileTime>;

    using JacobiType =
        Eigen::Matrix<Real, FunctionSet::OutputType::SizeAtCompileTime,
                      FunctionSet::InputType::SizeAtCompileTime>;

    auto InputVecToInputType = [](const InputVec &x) ->
        typename FunctionSet::InputType {
          return Eigen::Map<const typename FunctionSet::InputType>(x.data());
        };

    auto OutputTypeToOutputVec =
        [](const typename FunctionSet::OutputType &y) -> OutputVec {
      return Eigen::Map<const OutputVec>(y.data());
    };

    InputVec x = InputVec::Random();

    while (!f.ValidInput(InputVecToInputType(x))) {
      x = InputVec::Random();
    }

    Real eps = algebra::Eps<Real>();
    OutputVec y = OutputTypeToOutputVec(f(InputVecToInputType(x)));
    OutputVec y_device =
        OutputTypeToOutputVec(DeviceValue(f, InputVecToInputType(x)));

    for (int i = 0; i < y.size(); i++) {
      EXPECT_NEAR(y(i), y_device(i), fmax(fabs(sqrt(eps) * y(i)), sqrt(eps)));
    }

    JacobiType J = f.Jacobian(InputVecToInputType(x));
    JacobiType J_finite_diff;

    JacobiType J_device = DeviceJacobian(f, InputVecToInputType(x));

    for (int i = 0; i < J.size(); i++) {
      EXPECT_NEAR(J(i), J_device(i), fmax(fabs(sqrt(eps) * J(i)), sqrt(eps)));
    }

    J_finite_diff.setZero();
    for (int j = 0; j < x.size(); j++) {
      InputVec x_plus = x;
      x_plus[j] += eps;
      OutputVec y_plus = OutputTypeToOutputVec(f(InputVecToInputType(x_plus)));

      InputVec x_minus = x;
      x_minus[j] -= eps;
      OutputVec y_minus =
          OutputTypeToOutputVec(f(InputVecToInputType(x_minus)));

      OutputVec dy = (y_plus - y_minus) / (2 * eps);

      J_finite_diff.col(j) = dy;
    }

    // std::cout << std::fixed;
    // std::cout << "x:\n" << Eigen::Map<FunctionSet::InputType>(x.data()) <<
    // std::endl; std::cout << "y:\n" <<
    // Eigen::Map<FunctionSet::OutputType>(y.data()) << std::endl; std::cout <<
    // "J: \n" << J << std::endl; std::cout << "J_finite_diff: \n" <<
    // J_finite_diff << std::endl;

    // Compare J and J_finite_diff
    for (int j = 0; j < J.size(); j++) {
      EXPECT_NEAR(J(j), J_finite_diff(j),
                  fmax(fabs(sqrt(eps) * J(j)), sqrt(eps)));
    }

    using HessianType =
        HessianTensor<Real, FunctionSet::OutputType::SizeAtCompileTime,
                      FunctionSet::InputType::SizeAtCompileTime>;
    HessianType H = f.Hessian(InputVecToInputType(x));
    HessianType H_finite_diff;

    HessianType H_device = DeviceHessian(f, InputVecToInputType(x));

    for (int j = 0; j < OutputVec::SizeAtCompileTime; j++) {
      for (int k = 0; k < InputVec::SizeAtCompileTime; k++) {
        for (int l = 0; l < InputVec::SizeAtCompileTime; l++) {
          EXPECT_NEAR(H.m[j](k, l), H_device.m[j](k, l),
                      fmax(fabs(sqrt(eps) * H.m[j](k, l)), sqrt(eps)));
        }
      }
    }

    for (int j = 0; j < x.size(); j++) {
      InputVec x_plus = x;
      x_plus[j] += eps;
      JacobiType J_plus = f.Jacobian(InputVecToInputType(x_plus));

      InputVec x_minus = x;
      x_minus[j] -= eps;
      JacobiType J_minus = f.Jacobian(InputVecToInputType(x_minus));

      JacobiType dJ = (J_plus - J_minus) / (2 * eps);

      for (int k = 0; k < dJ.rows(); k++) {
        for (int l = 0; l < dJ.cols(); l++) {
          H_finite_diff.m[k](j, l) = dJ(k, l);
        }
      }
    }

    // std::cout << std::fixed;
    // std::cout << "x:\n" << x << std::endl;
    // std::cout << "H: \n" << H << std::endl;
    // std::cout << "H_finite_diff: \n" << H_finite_diff << std::endl;
    // std::cout << "Diff: \n" << H - H_finite_diff << std::endl;
    // std::cout.flush();

    for (int j = 0; j < OutputVec::SizeAtCompileTime; j++) {
      for (int k = 0; k < InputVec::SizeAtCompileTime; k++) {
        for (int l = 0; l < InputVec::SizeAtCompileTime; l++) {
          EXPECT_NEAR(H.m[j](k, l), H_finite_diff.m[j](k, l),
                      fmax(fabs(sqrt(eps) * H.m[j](k, l)), sqrt(eps)));
        }
      }
    }
  }
}
TEST(Physics, FunctionDerivativeDeterminant3) {
  TestFunctionSet<Determinant3<double>>();
}

TEST(Physics, FunctionDerivativeLogDeterminant3) {
  TestFunctionSet<LogDeterminant3<double>>();
}

TEST(Physics, FunctionDerivativeLogSquareDeterminant3) {
  TestFunctionSet<LogSquareDeterminant3<double>>();
}

TEST(Physics, FunctionDerivativeVecLength) {
  TestFunctionSet<VecLength<double, 3>>();
  TestFunctionSet<VecLength<double, 4>>();
  TestFunctionSet<VecLength<double, 5>>();
}

TEST(Physics, FunctionDerivativeVecNormalized) {
  TestFunctionSet<VecNormalized<double, 3>>();
  TestFunctionSet<VecNormalized<double, 4>>();
  TestFunctionSet<VecNormalized<double, 5>>();
}

TEST(Physics, FunctionDerivativeCross3) {
  TestFunctionSet<Cross3<double>>();
}

TEST(Physics, FunctionDerivativeDot) {
  TestFunctionSet<Dot<double>>();
}

TEST(Physics, FunctionDerivativeCrossNormalized) {
  TestFunctionSet<CrossNormalized<double>>();
}

TEST(Physics, FunctionDerivativeAtan2) {
  TestFunctionSet<Atan2<double>>();
}

TEST(Physics, FunctionDerivativeElasticNeoHookean) {
  TestFunctionSet<ElasticNeoHookean<double>>();
}

TEST(Physics, FunctionDerivativeDihedralAngleAssistEdgesToNormalsAxis) {
  TestFunctionSet<DihedralAngleAssistEdgesToNormalsAxis<double>>();
}

TEST(Physics, FunctionDerivativeDihedralAngleAssistNormalsAxisToSinCosTheta) {
  TestFunctionSet<DihedralAngleAssistNormalsAxisToSinCosTheta<double>>();
}

TEST(Physics, FunctionDerivativeDihedralAngleByEdges) {
  TestFunctionSet<DihedralAngleByEdges<double>>();
}

TEST(Physics, FunctionDerivativeDihedralAngleAssistVerticesToEdges) {
  TestFunctionSet<DihedralAngleAssistVerticesToEdges<double>>();
}

TEST(Physics, FunctionDerivativeDihedralAngleByVertices) {
  TestFunctionSet<DihedralAngleByVertices<double>>();
}

TEST(Physics, FunctionDerivativeDihedralEnergy) {
  DihedralEnergy<double> f;
  std::random_device rd;
  for (int i = 0; i < 100; i++) {
    f.rest_angle = std::uniform_real_distribution<double>(
        -glm::pi<double>() * 0.5, glm::pi<double>() * 0.5)(rd);
    TestFunctionSet(f, 1);
  }
}

TEST(Physics, FunctionDerivativeFEMTetrahedronDeformationGradient) {
  Eigen::Matrix3<double> Dm;
  do {
    Dm = Eigen::Matrix3<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<FEMTetrahedronDeformationGradient<double>>({Dm});
}

TEST(Physics, FunctionDerivativeFEMDeformationGradient3x2To3x3) {
  TestFunctionSet<FEMDeformationGradient3x2To3x3<double>>();
}

TEST(Physics, FunctionDerivativeFEMTriangleDeformationGradient3x2) {
  Eigen::Matrix2<double> Dm;
  do {
    Dm = Eigen::Matrix2<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<FEMTriangleDeformationGradient3x2<double>>({Dm});
}

TEST(Physics, FunctionDerivativeFEMTriangleDeformationGradient3x3) {
  Eigen::Matrix2<double> Dm;
  do {
    Dm = Eigen::Matrix2<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<FEMTriangleDeformationGradient3x3<double>>({Dm});
}
